#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
#define MIN_VALUE (-1e38)
typedef at::BFloat16 bf16;

__global__ void kernel_forward(const int B, const int T, const int C,
                               const float *__restrict__ const _w, const bf16 *__restrict__ const _u, const bf16 *__restrict__ const _k, const bf16 *__restrict__ const _v,
                               const float *__restrict__ const last_state, bf16 *__restrict__ const _y, float *__restrict__ const new_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset = (_b * C + _c)*3;

    float u = float(_u[_c]);
    float w = _w[_c];
    const bf16 *__restrict__ const k = _k + _offset;
    const bf16 *__restrict__ const v = _v + _offset;
    bf16 *__restrict__ const y = _y + _offset;

    float p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    // p and q are running sums divided by exp(o) (to avoid overflows)
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);

        float no = max(o, u + kk);
        float A = exp(o - no);
        float B = exp(u + kk - no);
        y[ii] = bf16((A * p + B * vv) / (A * q + B));

        no = max(w + o, kk);
        A = exp(w + o - no);
        B = exp(kk - no);
        p = A * p + B * vv;
        q = A * q + B;
        o = no;
    }
    if (new_state != NULL) {
        new_state[state_offset+0] = p;
        new_state[state_offset+1] = q;
        new_state[state_offset+2] = o;
    }
}

__global__ void kernel_backward(const int B, const int T, const int C,
                                const float *__restrict__ const _w, const bf16 *__restrict__ const _u, const bf16 *__restrict__ const _k, const bf16 *__restrict__ const _v, const float *__restrict__ const last_state, 
                                const bf16 *__restrict__ const _gy, const float *__restrict__ const gnew_state,
                                bf16 *__restrict__ const _gw, bf16 *__restrict__ const _gu, bf16 *__restrict__ const _gk, bf16 *__restrict__ const _gv, float *__restrict__ const glast_state) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int _b = idx / C;
    const int _c = idx % C;
    const int _offset = _b * T * C + _c;
    const int state_offset  = (_b * C + _c)*3;

    float u = float(_u[_c]);
    float w = _w[_c];
    const bf16 *__restrict__ const k = _k + _offset;
    const bf16 *__restrict__ const v = _v + _offset;
    const bf16 *__restrict__ const gy = _gy + _offset;

    bf16 *__restrict__ const gk = _gk + _offset;
    bf16 *__restrict__ const gv = _gv + _offset;

    float y[Tmax], z[Tmax], zexp[Tmax];

    float gw = 0, gu = 0;
    float dpdw = 0, dqdw = 0;
    float p, q, o;
    if (last_state == NULL) {
        p = 0, q = 0, o = MIN_VALUE;
    } else {
        p = last_state[state_offset+0];
        q = last_state[state_offset+1];
        o = last_state[state_offset+2];
    }
    for (int i = 0; i < T; i++) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float gyy = float(gy[ii]);

        float no = max(o, kk + u);
        float A = exp(o - no);
        float B = exp(kk + u - no);

        float num = A * p + B * vv;
        float iden = 1 / (A * q + B);

        y[i] = num * iden;
        z[i] = iden;
        zexp[i] = kk + u - no;

        gw += gyy * (dpdw - dqdw * y[i]) * iden * A;
        gu += gyy * (vv - y[i]) * B * iden;

        no = max(w + o, kk);
        A = exp(w + o - no);
        B = exp(kk - no);
        dpdw = A * (p + dpdw);
        dqdw = A * (q + dqdw);
        p = A * p + B * vv;
        q = A * q + B;
        o = no;
    }

    float gp = 0, gq = 0, go = MIN_VALUE;
    if (gnew_state != NULL) {
        gp = gnew_state[state_offset+0];
        gq = gnew_state[state_offset+1];
        go = gnew_state[state_offset+2];
        if (gp == 0 && gq == 0) go = MIN_VALUE;
        gw += (gp * dpdw + gq * dqdw) * exp(o+go);
    }

    for (int i = T - 1; i >= 0; i--) {
        const int ii = i * C;
        const float kk = float(k[ii]);
        const float vv = float(v[ii]);
        const float gyy = float(gy[ii]);

        float A = gyy * z[i] * exp(zexp[i]);
        float B = exp(kk + go);
        gk[ii] = bf16(A * (vv - y[i]) + B * (gp * vv + gq));
        gv[ii] = bf16(A + B * gp);

        float no = max(w + go, zexp[i] - kk - u);
        A = exp(w + go - no);
        B = gyy * z[i] * exp(zexp[i] - kk - u - no);
        gp = A * gp + B;
        gq = A * gq - B * y[i];
        go = no;
    }

    // glast_state[2] is not the gradient w.r.t of last_state[2]
    // o (index 2) in last_state is just an exponent for p and q
    // so there are really only 2 elements to differentiate on
    // Similary go (glast_state index 2) is just an exponent for gp and gq
    if (glast_state != NULL) {
        glast_state[state_offset+0] = gp;
        glast_state[state_offset+1] = gq;
        glast_state[state_offset+2] = go;
    }

    // Multiply by w because the w -> -exp(w) preprocessing is halfway in the backwards pass, even though it's not in the forward pass
    const int _offsetBC = _b * C + _c;
    _gw[_offsetBC] = bf16(gw * _w[_c]);
    _gu[_offsetBC] = gu;
}

void cuda_forward(int B, int T, int C, float *w, bf16 *u, bf16 *k, bf16 *v, float *last_state, bf16 *y, float *new_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_forward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, y, new_state);
}

void cuda_backward(int B, int T, int C, float *w, bf16 *u, bf16 *k, bf16 *v, float *last_state, bf16 *gy, float *gnew_state, bf16 *gw, bf16 *gu, bf16 *gk, bf16 *gv, float *glast_state) {
    dim3 threadsPerBlock( min(C, 32) ); // requires --maxrregcount 60 for optimal performance
    assert(B * C % threadsPerBlock.x == 0);
    dim3 numBlocks(B * C / threadsPerBlock.x);
    kernel_backward<<<numBlocks, threadsPerBlock>>>(B, T, C, w, u, k, v, last_state, gy, gnew_state, gw, gu, gk, gv, glast_state);
}
